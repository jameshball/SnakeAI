//#ifdef __HIPCC__

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "matrix.h"

#define HOST_ELEM(m, row, col) ((m)->hostM[(row) + (col) * (m)->rows])
#define DEVICE_ELEM(m, row, col) ((m)->devM[(row) + (col) * (m)->rows])

#define size(m) m->rows * m->cols
#define ELEM_WISE(m, i, op)           \
  for (int i = 0; i < size(m); i++) { \
    op;                               \
  }

struct matrix_t {
  int rows;
  int cols;

  bool dirtyHost;
  float *hostM;
  bool dirtyDevice;
  float *devM;

  hipblasHandle_t cublasH;
};

static inline hipblasStatus_t host_to_dev(matrix_t *m){
  assert(!m->dirtyHost);

  hipblasStatus_t stat = hipblasSetMatrix(m->rows, m->cols, sizeof(float), m->hostM, m->rows, m->devM, m->rows);
  if (stat == HIPBLAS_STATUS_SUCCESS) {
    m->dirtyDevice = false;
  } else {
    printf("host to device transfer failed");
  }
  return stat;
}

static inline hipblasStatus_t dev_to_host(matrix_t *m){
  assert(!m->dirtyDevice);

  hipblasStatus_t stat = hipblasGetMatrix(m->rows, m->cols, sizeof(float), m->devM, m->rows, m->hostM, m->rows);
  if (stat == HIPBLAS_STATUS_SUCCESS) {
    m->dirtyHost = false;
  } else {
    printf("device to host transfer failed");
  }
  return stat;
}

static inline void sync_host(matrix_t *m){
  if (m->dirtyHost){
    dev_to_host(m);
  }
}

static inline void sync_dev(matrix_t *m){
  if (m->dirtyDevice){
    host_to_dev(m);
  }
}

matrix_t *init_matrix(int rows, int cols){
  float *hostM =(float*) calloc(rows * cols, sizeof(float));
  if (!hostM){
    printf("RAM allocation failed\n");
    return NULL;
  }

  float* devM;
  hipError_t cudaStat = hipMalloc((void **)&devM, rows * cols * sizeof(float));
  if (cudaStat != hipSuccess){
    printf("device allocation failed\n");
    return NULL;
  }

  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS){
    printf("cublas initialisation failed\n");
    return NULL;
  }

  stat = hipblasSetMatrix(rows, cols, sizeof(float), hostM, rows, devM, rows);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("host to device transfer failed at %d", __LINE__);
    hipFree(devM);
    hipblasDestroy(handle);
    return NULL;
  }

  matrix_t *m = (matrix_t*) malloc(sizeof(matrix_t));

  m->rows = rows;
  m->cols = cols;
  m->hostM = hostM;
  m->devM = devM;
  m->dirtyHost = false;
  m->dirtyDevice = false;

  m->cublasH = handle;
  
  return m;
}

void matrix_fill(matrix_t *m, float x){
  ELEM_WISE(m,i,m->hostM[i] = x);
}



//#endif
